#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#define DATA_SIZE 10


template <class R, class S, class T>	
	T hypothesis_test(R hypothesis_value, T alpha, S test_type){
	thrust::host_vector<T> data(DATA_SIZE); 
	for (int i=0; i < data.size(); i++){
		data[i] = i + 1;
	}	

	thrust::device_vector<T> D1 = data;
	thrust::device_vector<T> D2(data.size());
	
	T mean = thrust::reduce(	D1.begin(), 
					D1.end(), 
					(T)0, 
					thrust::plus<T>())/data.size();

	thrust::transform(	D1.begin(), 
				D1.end(), 
				thrust::make_constant_iterator(mean), 
				D1.begin(), 
				thrust::minus<T>());

	thrust::transform(D1.begin(), D1.end(), D1.begin(), D2.begin(), thrust::multiplies<T>());
	T variance = thrust::reduce(D2.begin(), D2.end(),(T)0, thrust::plus<T>())/(data.size()-1);
	
	T standard_deviation = sqrt(variance);
	T Z = (mean - hypothesis_value)/(standard_deviation/sqrt(data.size()));
		
	T left = (0.5)*(1.0 + erf(Z/sqrt(2.0)));
	T right = 1.0 - (0.5)*(1.0 + erf(Z/sqrt(2.0)));
	T two_sided = 2.0*(1.0 - (0.5)*(1.0 + erf(abs(Z)/sqrt(2.0))));
 //Something is wrong with this equation. Two sided allots half of our alpha to each side...	

	if (test_type == 1){
		if(left < alpha){
			printf("We reject the null hypothesis\n");
				return left; 	 
			}
			else{	
				printf("We fail to reject the null hypothesis\n");
					return left;
			}
	}
	else if(test_type == 2){
		if(right < alpha){
			printf("We reject the null hypothesis \n");
				return right;
			}
			else{
				printf("We fail to reject the null hypothesis \n");
					return right;
			}
	}
	else if(test_type == 3){
		if(two_sided < alpha){
			printf("We reject the null hypothesis \n");
				return two_sided;
			}
			else{
				printf("we fail to reject the null hypothesis \n");
					return two_sided;
			}
	}
	else return 0;
	

}	
	
int main(){

	float value = hypothesis_test(4,0.05, 3);
	printf("%.4f \n", value);
 
		
}
	





