#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_real_distribution.h> 
#include <thrust/random.h>
#include <iostream>
#define DATA_SIZE 100000000


struct psrngen
{
	__host__ __device__ psrngen(float _a, float _b) : a(_a), b(_b) {;}
	__host__ __device__ float operator()(const unsigned int n) const
	{
		thrust::default_random_engine rng;
		thrust::uniform_real_distribution<float> dist(a, b);
		rng.discard(n);
		return dist(rng);
	}
	float a, b;

};


template <class R, class S, class T>	
T hypothesis_test(R hypothesis_value, T alpha, S test_type){
	
	srand(time(NULL));
	int _seed = rand();
	thrust::device_vector<float> D1(DATA_SIZE);
	thrust::counting_iterator<unsigned int> index_sequence_begin(_seed);
	thrust::transform(	index_sequence_begin, 
				index_sequence_begin + (DATA_SIZE), 
				D1.begin(), 
				psrngen(0.0, 1.0f));
	thrust::device_vector<T> D2(D1.size());


//	for(int j=0; j <D1.size();j++){
//		std::cout <<"D1["<< j <<"] = " << D1[j] << std::endl;
//	}


	T mean = thrust::reduce(	D1.begin(), 
					D1.end(), 
					(T)0, 
					thrust::plus<T>())/D1.size();


	thrust::transform(		D1.begin(), 
					D1.end(), 
					thrust::make_constant_iterator(mean), 
					D1.begin(), 
					thrust::minus<T>());

	thrust::transform(		D1.begin(), 
					D1.end(), 
					D1.begin(), 
					D2.begin(), 
					thrust::multiplies<T>());

	T variance = thrust::reduce(D2.begin(), D2.end(),(T)0, thrust::plus<T>())/(D1.size()-1);
	

	T standard_deviation = sqrt(variance);
	T Z = (mean - hypothesis_value)/(standard_deviation/sqrt(D1.size()));

		
	T left = (0.5)*(1.0 + erf(Z/sqrt(2.0)));
	T right = 1.0 - (0.5)*(1.0 + erf(Z/sqrt(2.0)));
	T two_sided = 2.0*(1.0 - (0.5)*(1.0 + erf(abs(Z)/sqrt(2.0))));
	

	if (test_type == 1){
		if(left < alpha){
			printf("We reject the null hypothesis\n");
				return left; 	 
		}
		else{	
			printf("We fail to reject the null hypothesis\n");
				return left;
		}
	}

	else if(test_type == 2){
		if(right < alpha){
			printf("We reject the null hypothesis \n");
				return right;
		}
		else{
			printf("We fail to reject the null hypothesis \n");
				return right;
		}
	}

	else if(test_type == 3){
		if(two_sided < alpha){
			printf("We reject the null hypothesis \n");
				return two_sided;
		}
		else{
			printf("we fail to reject the null hypothesis \n");
				return two_sided;
		}
	}

	else return 0;
	

}	
	
int main(){

	float value = hypothesis_test(0.5,0.05, 3);
	printf("%.4f \n", value);
 
		
}
	





